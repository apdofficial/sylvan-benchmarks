#include "utility.h"

#include <hip/hip_runtime.h>

size_t getFreeCudaMemory() {
	size_t freeMemory;
	size_t totalMemory;
	hipMemGetInfo(&freeMemory, &totalMemory);

	return freeMemory;
}

size_t getTotalCudaMemory() {
	size_t freeMemory;
	size_t totalMemory;
	hipMemGetInfo(&freeMemory, &totalMemory);

	return totalMemory;
}

bool resetCudaDevice() {
	hipError_t result = hipDeviceReset();
	return (result == hipSuccess);
}

int getRuntimeCudaVersion() {
	int result = -1;
	hipError_t errorResult = hipRuntimeGetVersion(&result);
	if (errorResult != hipSuccess) {
		return -1;
	}
	return result;
}