
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void cuda_kernel_kernelSwitchTest(int const * const A, int * const B) {
	*B = *A;
}

void kernelSwitchTest(size_t N) {
	int* deviceIntA;
	int* deviceIntB;

	if (hipMalloc((void**)&deviceIntA, sizeof(int)) != hipSuccess) {
		std::cout << "Error in hipMalloc while allocating " << sizeof(int) << " Bytes!" << std::endl;
		return;
	}
	if (hipMalloc((void**)&deviceIntB, sizeof(int)) != hipSuccess) {
		std::cout << "Error in hipMalloc while allocating " << sizeof(int) << " Bytes!" << std::endl;
		return;
	}

	// Allocate space on the device
	auto start_time = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < N; ++i) {
		cuda_kernel_kernelSwitchTest<<<1,1>>>(deviceIntA, deviceIntB);
	}
	auto end_time = std::chrono::high_resolution_clock::now();
	std::cout << "Switching the Kernel " << N << " times took " << std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() << "micros" << std::endl;
	std::cout << "Resulting in " << (std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / ((double)(N))) << "Microseconds per Kernel Switch" << std::endl;

	// Free memory on device
	if (hipFree(deviceIntA) != hipSuccess) {
		std::cout << "Error in cudaFree!" << std::endl;
		return;
	}
	if (hipFree(deviceIntB) != hipSuccess) {
		std::cout << "Error in cudaFree!" << std::endl;
		return;
	}
}