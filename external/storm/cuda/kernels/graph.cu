#include "hip/hip_runtime.h"
#include <stdio.h>
 
const int N = 16; 
const int blocksize = 16;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ 
void hello(char *a, int *b) 
{
    a[threadIdx.x] += b[threadIdx.x];
}

namespace stormcuda {
    namespace graph {
        void helloWorld() {
            printf("CUDA TEST START\n");
            printf("Should print \"Hello World\"\n");

            char a[N] = "Hello \0\0\0\0\0\0";
            int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            char c[N] = "YELLO \0\0\0\0\0\0";

            char *ad;
            int *bd;
            const int csize = N * sizeof(char);
            const int isize = N * sizeof(int);

            printf("%s", a);

            hipMalloc((void **) &ad, csize);
            hipMalloc((void **) &bd, isize);
            hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
            hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

            dim3 dimBlock(blocksize, 1);
            dim3 dimGrid(1, 1);
            hello << < dimGrid, dimBlock >> > (ad, bd);

            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());

            hipMemcpy(c, ad, csize, hipMemcpyDeviceToHost);
            hipFree(ad);
            hipFree(bd);

            printf("%s\n", c);
            printf("CUDA TEST END\n");
        }
    }
}
