#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include <chrono>
#include <iostream>

__global__ void cuda_kernel_basicAdd(int a, int b, int *c) { 
	*c = a + b; 
}

__global__ void cuda_kernel_arrayFma(int const * const A, int const * const B, int const * const C, int * const D, int const N) {
	// Fused Multiply Add:
	// A * B + C => D

	/*
     *Die Variable i dient f�r den Zugriff auf das Array. Da jeder Thread die Funktion VecAdd
     *ausf�hrt, muss i f�r jeden Thread unterschiedlich sein. Ansonsten w�rden unterschiedliche
     *Threads auf denselben Index im Array schreiben. blockDim.x ist die Anzahl der Threads der x-Komponente
     *des Blocks, blockIdx.x ist die x-Koordinate des aktuellen Blocks und threadIdx.x ist die x-Koordinate des
     *Threads, der die Funktion gerade ausf�hrt.
    */
    int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N) {
		D[i] = A[i] * B[i] + C[i];
	}
}

__global__ void cuda_kernel_arrayFmaOptimized(int * const A, int const N, int const M) {
	// Fused Multiply Add:
	// A * B + C => D

	// Layout:
	// A B C D A B C D A B C D

    int i = blockDim.x * blockIdx.x + threadIdx.x;

	if ((i*M) < N) {
		for (int j = i*M; j < i*M + M; ++j) {
			A[j*4 + 3] = A[j*4] * A[j*4 + 1] + A[j*4 + 2];
		}
	}
}

extern "C" int cuda_basicAdd(int a, int b) {
	int c = 0;
	int *dev_c;
	hipMalloc((void**)&dev_c, sizeof(int));
	cuda_kernel_basicAdd<<<1, 1>>>(a, b, dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	//printf("%d + %d + 42 is %d\n", a, b, c);
	hipFree(dev_c);
	return c;
}

void cpp_cuda_bandwidthTest(int entryCount, int N) {
	// Size of the Arrays
	size_t arraySize = entryCount * sizeof(int);
	
	int* deviceIntArray;
	int* hostIntArray = new int[arraySize];

	// Allocate space on the device
	auto start_time = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < N; ++i) {
		if (hipMalloc((void**)&deviceIntArray, arraySize) != hipSuccess) {
			std::cout << "Error in hipMalloc while allocating " << arraySize << " Bytes!" << std::endl;
			delete[] hostIntArray;
			return;
		}
		// Free memory on device
		if (hipFree(deviceIntArray) != hipSuccess) {
			std::cout << "Error in cudaFree!" << std::endl;
			delete[] hostIntArray;
			return;
		}
	}
	auto end_time = std::chrono::high_resolution_clock::now();
	auto copyTime = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count();
	double mBytesPerSecond = (((double)(N * arraySize)) / copyTime) * 0.95367431640625;
	std::cout << "Allocating the Array " << N << " times took " << copyTime << " Microseconds." << std::endl;
	std::cout << "Resulting in " << mBytesPerSecond << " MBytes per Second Allocationspeed." << std::endl;

	if (hipMalloc((void**)&deviceIntArray, arraySize) != hipSuccess) {
		std::cout << "Error in hipMalloc while allocating " << arraySize << " Bytes for copyTest!" << std::endl;
		delete[] hostIntArray;
		return;
	}
	
	// Prepare data
	for (int i = 0; i < N; ++i) {
		hostIntArray[i] = i * 333 + 123;
	}

	// Copy data TO device
	start_time = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < N; ++i) {
		if (hipMemcpy(deviceIntArray, hostIntArray, arraySize, hipMemcpyHostToDevice) != hipSuccess) {
			std::cout << "Error in hipMemcpy while copying " << arraySize << " Bytes to device!" << std::endl;
			// Free memory on device
			if (hipFree(deviceIntArray) != hipSuccess) {
				std::cout << "Error in cudaFree!" << std::endl;
			}
			delete[] hostIntArray;
			return;
		}
	}
	end_time = std::chrono::high_resolution_clock::now();
	copyTime = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count();
	mBytesPerSecond = (((double)(N * arraySize)) / copyTime) * 0.95367431640625;
	std::cout << "Copying the Array " << N << " times took " << copyTime << " Microseconds." << std::endl;
	std::cout << "Resulting in " << mBytesPerSecond << " MBytes per Second TO device." << std::endl;

	// Copy data FROM device
	start_time = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < N; ++i) {
		if (hipMemcpy(hostIntArray, deviceIntArray, arraySize, hipMemcpyDeviceToHost) != hipSuccess) {
			std::cout << "Error in hipMemcpy while copying " << arraySize << " Bytes to host!" << std::endl;
			// Free memory on device
			if (hipFree(deviceIntArray) != hipSuccess) {
				std::cout << "Error in cudaFree!" << std::endl;
			}
			delete[] hostIntArray;
			return;
		}
	}
	end_time = std::chrono::high_resolution_clock::now();
	copyTime = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count();
	mBytesPerSecond = (((double)(N * arraySize)) / copyTime) * 0.95367431640625;
	std::cout << "Copying the Array " << N << " times took " << copyTime << " Microseconds." << std::endl;
	std::cout << "Resulting in " << mBytesPerSecond << " MBytes per Second FROM device." << std::endl;

	// Free memory on device
	if (hipFree(deviceIntArray) != hipSuccess) {
		std::cout << "Error in cudaFree!" << std::endl;
	}
	delete[] hostIntArray;
}

extern "C" void cuda_arrayFma(int const * const A, int const * const B, int const * const C, int * const D, int const N) {
	// Size of the Arrays
	size_t arraySize = N * sizeof(int);
	
	int* deviceIntArrayA;
	int* deviceIntArrayB;
	int* deviceIntArrayC;
	int* deviceIntArrayD;

	// Allocate space on the device
	if (hipMalloc((void**)&deviceIntArrayA, arraySize) != hipSuccess) {
		printf("Error in cudaMalloc1!\n");
		return;
	}
	if (hipMalloc((void**)&deviceIntArrayB, arraySize) != hipSuccess) {
		printf("Error in cudaMalloc2!\n");
		hipFree(deviceIntArrayA);
		return;
	}
	if (hipMalloc((void**)&deviceIntArrayC, arraySize) != hipSuccess) {
		printf("Error in cudaMalloc3!\n");
		hipFree(deviceIntArrayA);
		hipFree(deviceIntArrayB);
		return;
	}
	if (hipMalloc((void**)&deviceIntArrayD, arraySize) != hipSuccess) {
		printf("Error in cudaMalloc4!\n");
		hipFree(deviceIntArrayA);
		hipFree(deviceIntArrayB);
		hipFree(deviceIntArrayC);
		return;
	}
	
	// Copy data TO device
	if (hipMemcpy(deviceIntArrayA, A, arraySize, hipMemcpyHostToDevice) != hipSuccess) {
		printf("Error in cudaMemcpy!\n");
		hipFree(deviceIntArrayA);
		hipFree(deviceIntArrayB);
		hipFree(deviceIntArrayC);
		hipFree(deviceIntArrayD);
		return;
	}
	if (hipMemcpy(deviceIntArrayB, B, arraySize, hipMemcpyHostToDevice) != hipSuccess) {
		printf("Error in cudaMemcpy!\n");
		hipFree(deviceIntArrayA);
		hipFree(deviceIntArrayB);
		hipFree(deviceIntArrayC);
		hipFree(deviceIntArrayD);
		return;
	}
	if (hipMemcpy(deviceIntArrayC, C, arraySize, hipMemcpyHostToDevice) != hipSuccess) {
		printf("Error in cudaMemcpy!\n");
		hipFree(deviceIntArrayA);
		hipFree(deviceIntArrayB);
		hipFree(deviceIntArrayC);
		hipFree(deviceIntArrayD);
		return;
	}
	
    // Festlegung der Threads pro Block
    int threadsPerBlock = 512;
    // Es werden soviele Bl�cke ben�tigt, dass alle Elemente der Vektoren abgearbeitet werden k�nnen
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	// Run kernel
	cuda_kernel_arrayFma<<<blocksPerGrid, threadsPerBlock>>>(deviceIntArrayA, deviceIntArrayB, deviceIntArrayC, deviceIntArrayD, N);

	// Copy data FROM device
	if (hipMemcpy(D, deviceIntArrayD, arraySize, hipMemcpyDeviceToHost) != hipSuccess) {
		printf("Error in cudaMemcpy!\n");
		hipFree(deviceIntArrayA);
		hipFree(deviceIntArrayB);
		hipFree(deviceIntArrayC);
		hipFree(deviceIntArrayD);
		return;
	}

	// Free memory on device
	hipFree(deviceIntArrayA);
	hipFree(deviceIntArrayB);
	hipFree(deviceIntArrayC);
	hipFree(deviceIntArrayD);
}

extern "C" void cuda_arrayFmaOptimized(int * const A, int const N, int const M) {
	// Size of the Arrays
	size_t arraySize = N * sizeof(int) * 4;
	
	int* deviceIntArrayA;

	// Allocate space on the device
	if (hipMalloc((void**)&deviceIntArrayA, arraySize) != hipSuccess) {
		printf("Error in cudaMalloc1!\n");
		return;
	}

#define ONFAILFREE0() do { } while(0)
#define ONFAILFREE1(a) do { hipFree(a); } while(0)
#define ONFAILFREE2(a, b) do { hipFree(a); hipFree(b); } while(0)
#define ONFAILFREE3(a, b, c) do { hipFree(a); hipFree(b); hipFree(c); } while(0)
#define ONFAILFREE4(a, b, c, d) do { hipFree(a); hipFree(b); hipFree(c); hipFree(d); } while(0)
#define CHECKED_CUDA_CALL(func__, freeArgs, ...) do { int retCode = cuda##func__ (__VA_ARGS__); if (retCode != hipSuccess) { freeArgs; printf("Error in func__!\n"); return; } } while(0)

	// Copy data TO device

	CHECKED_CUDA_CALL(Memcpy, ONFAILFREE1(deviceIntArrayA), deviceIntArrayA, A, arraySize, hipMemcpyHostToDevice);

	/*if (cudaMemcpy(deviceIntArrayA, A, arraySize, cudaMemcpyHostToDevice) != cudaSuccess) {
		printf("Error in cudaMemcpy!\n");
		cudaFree(deviceIntArrayA);
		return;
	}*/
	
    // Festlegung der Threads pro Block
    int threadsPerBlock = 512;
    // Es werden soviele Bl�cke ben�tigt, dass alle Elemente der Vektoren abgearbeitet werden k�nnen
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	// Run kernel
	cuda_kernel_arrayFmaOptimized<<<blocksPerGrid, threadsPerBlock>>>(deviceIntArrayA, N, M);

	// Copy data FROM device
	if (hipMemcpy(A, deviceIntArrayA, arraySize, hipMemcpyDeviceToHost) != hipSuccess) {
		printf("Error in cudaMemcpy!\n");
		hipFree(deviceIntArrayA);
		return;
	}

	// Free memory on device
	if (hipFree(deviceIntArrayA) != hipSuccess) {
		printf("Error in cudaFree!\n");
		return;
	}
}

extern "C" void cuda_arrayFmaHelper(int const * const A, int const * const B, int const * const C, int * const D, int const N) {
	for (int i = 0; i < N; ++i) {
		D[i] = A[i] * B[i] + C[i];
	}
}

extern "C" void cuda_arrayFmaOptimizedHelper(int * const A, int const N) {
	for (int i = 0; i < N; i += 4) {
		A[i+3] = A[i] * A[i+1] + A[i+2];
	}
}