#include "hip/hip_runtime.h"
#include "basicValueIteration.h"
#define CUSP_USE_TEXTURE_MEMORY

#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>
#include "hipsparse.h"

#include "utility.h"

#include "cuspExtension.h"

#include <thrust/transform.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>

#include "storm-cudaplugin-config.h"

#ifdef DEBUG
#define CUDA_CHECK_ALL_ERRORS() do { hipError_t errSync  = hipGetLastError(); hipError_t errAsync = hipDeviceSynchronize();	if (errSync != hipSuccess) { std::cout << "(DLL) Sync kernel error: " << hipGetErrorString(errSync) << " (Code: " << errSync << ") in Line " << __LINE__ << std::endl; } if (errAsync != hipSuccess) { std::cout << "(DLL) Async kernel error: " << hipGetErrorString(errAsync) << " (Code: " << errAsync << ") in Line " << __LINE__ << std::endl; } } while(false)
#else
#define CUDA_CHECK_ALL_ERRORS() do {} while (false)
#endif

template<typename T, bool Relative>
struct equalModuloPrecision : public thrust::binary_function<T,T,T>
{
__host__ __device__ T operator()(const T &x, const T &y) const
{
    if (Relative) {
		if (y == 0) {
			return ((x >= 0) ? (x) : (-x));
		}
		const T result = (x - y) / y;
		return ((result >= 0) ? (result) : (-result));
    } else {
        const T result = (x - y);
		return ((result >= 0) ? (result) : (-result));
    }
}
};

template<typename IndexType, typename ValueType>
void exploadVector(std::vector<std::pair<IndexType, ValueType>> const& inputVector, std::vector<IndexType>& indexVector, std::vector<ValueType>& valueVector) {
	indexVector.reserve(inputVector.size());
	valueVector.reserve(inputVector.size());
	for (size_t i = 0; i < inputVector.size(); ++i) {
		indexVector.push_back(inputVector.at(i).first);
		valueVector.push_back(inputVector.at(i).second);
	}
}

// TEMPLATE VERSION
template <bool Minimize, bool Relative, typename IndexType, typename ValueType>
bool basicValueIteration_mvReduce(uint_fast64_t const maxIterationCount, double const precision, std::vector<IndexType> const& matrixRowIndices, std::vector<storm::storage::MatrixEntry<uint_fast64_t, ValueType>> const& columnIndicesAndValues, std::vector<ValueType>& x, std::vector<ValueType> const& b, std::vector<IndexType> const& nondeterministicChoiceIndices, size_t& iterationCount) {
	//std::vector<IndexType> matrixColumnIndices;
	//std::vector<ValueType> matrixValues;
	//exploadVector<IndexType, ValueType>(columnIndicesAndValues, matrixColumnIndices, matrixValues);
	bool errorOccured = false;

	IndexType* device_matrixRowIndices = nullptr;
	ValueType* device_matrixColIndicesAndValues = nullptr;
	ValueType* device_x = nullptr;
	ValueType* device_xSwap = nullptr;
	ValueType* device_b = nullptr;
	ValueType* device_multiplyResult = nullptr;
	IndexType* device_nondeterministicChoiceIndices = nullptr;

#ifdef DEBUG
	std::cout.sync_with_stdio(true);
	std::cout << "(DLL) Entering CUDA Function: basicValueIteration_mvReduce" << std::endl;
	std::cout << "(DLL) Device has " << getTotalCudaMemory() << " Bytes of Memory with " << getFreeCudaMemory() << "Bytes free (" << (static_cast<double>(getFreeCudaMemory()) / static_cast<double>(getTotalCudaMemory())) * 100 << "%)." << std::endl;
	size_t memSize = sizeof(IndexType) * matrixRowIndices.size() + sizeof(IndexType) * columnIndicesAndValues.size() * 2 + sizeof(ValueType) * x.size() + sizeof(ValueType) * x.size() + sizeof(ValueType) * b.size() + sizeof(ValueType) * b.size() + sizeof(IndexType) * nondeterministicChoiceIndices.size();
	std::cout << "(DLL) We will allocate " << memSize << " Bytes." << std::endl;
#endif

	const IndexType matrixRowCount = matrixRowIndices.size() - 1;
	const IndexType matrixColCount = nondeterministicChoiceIndices.size() - 1;
	const IndexType matrixNnzCount = columnIndicesAndValues.size();

	hipError_t cudaMallocResult;

	bool converged = false;
	iterationCount = 0;

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_matrixRowIndices), sizeof(IndexType) * (matrixRowCount + 1));
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Matrix Row Indices, Error Code " << cudaMallocResult << "." << std::endl;
		errorOccured = true;
		goto cleanup;
	}

#ifdef STORM_CUDAPLUGIN_HAVE_64BIT_FLOAT_ALIGNMENT
#define STORM_CUDAPLUGIN_HAVE_64BIT_FLOAT_ALIGNMENT_VALUE true
#else
#define STORM_CUDAPLUGIN_HAVE_64BIT_FLOAT_ALIGNMENT_VALUE false
#endif
	if (sizeof(ValueType) == sizeof(float) && STORM_CUDAPLUGIN_HAVE_64BIT_FLOAT_ALIGNMENT_VALUE) {
		CUDA_CHECK_ALL_ERRORS();
		cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_matrixColIndicesAndValues), sizeof(IndexType) * matrixNnzCount + sizeof(IndexType) * matrixNnzCount);
		if (cudaMallocResult != hipSuccess) {
			std::cout << "Could not allocate memory for Matrix Column Indices and Values, Error Code " << cudaMallocResult << "." << std::endl;
			errorOccured = true;
			goto cleanup;
		}
	} else {
		CUDA_CHECK_ALL_ERRORS();
		cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_matrixColIndicesAndValues), sizeof(IndexType) * matrixNnzCount + sizeof(ValueType) * matrixNnzCount);
		if (cudaMallocResult != hipSuccess) {
			std::cout << "Could not allocate memory for Matrix Column Indices and Values, Error Code " << cudaMallocResult << "." << std::endl;
			errorOccured = true;
			goto cleanup;
		}
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_x), sizeof(ValueType) * matrixColCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector x, Error Code " << cudaMallocResult << "." << std::endl;
		errorOccured = true;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_xSwap), sizeof(ValueType) * matrixColCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector x swap, Error Code " << cudaMallocResult << "." << std::endl;
		errorOccured = true;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_b), sizeof(ValueType) * matrixRowCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector b, Error Code " << cudaMallocResult << "." << std::endl;
		errorOccured = true;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_multiplyResult), sizeof(ValueType) * matrixRowCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector multiplyResult, Error Code " << cudaMallocResult << "." << std::endl;
		errorOccured = true;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_nondeterministicChoiceIndices), sizeof(IndexType) * (matrixColCount + 1));
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Nondeterministic Choice Indices, Error Code " << cudaMallocResult << "." << std::endl;
		errorOccured = true;
		goto cleanup;
	}

#ifdef DEBUG
	std::cout << "(DLL) Finished allocating memory." << std::endl;
#endif

	// Memory allocated, copy data to device
	hipError_t cudaCopyResult;

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_matrixRowIndices, matrixRowIndices.data(), sizeof(IndexType) * (matrixRowCount + 1), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Matrix Row Indices, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
	}

	// Copy all data as floats are expanded to 64bits :/
	if (sizeof(ValueType) == sizeof(float) && STORM_CUDAPLUGIN_HAVE_64BIT_FLOAT_ALIGNMENT_VALUE) {
		CUDA_CHECK_ALL_ERRORS();
		cudaCopyResult = hipMemcpy(device_matrixColIndicesAndValues, columnIndicesAndValues.data(), (sizeof(IndexType) * matrixNnzCount) + (sizeof(IndexType) * matrixNnzCount), hipMemcpyHostToDevice);
		if (cudaCopyResult != hipSuccess) {
			std::cout << "Could not copy data for Matrix Column Indices and Values, Error Code " << cudaCopyResult << std::endl;
			errorOccured = true;
			goto cleanup;
		}
	} else {
		CUDA_CHECK_ALL_ERRORS();
		cudaCopyResult = hipMemcpy(device_matrixColIndicesAndValues, columnIndicesAndValues.data(), (sizeof(IndexType) * matrixNnzCount) + (sizeof(ValueType) * matrixNnzCount), hipMemcpyHostToDevice);
		if (cudaCopyResult != hipSuccess) {
			std::cout << "Could not copy data for Matrix Column Indices and Values, Error Code " << cudaCopyResult << std::endl;
			errorOccured = true;
			goto cleanup;
		}
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_x, x.data(), sizeof(ValueType) * matrixColCount, hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector x, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
	}

	// Preset the xSwap to zeros...
	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemset(device_xSwap, 0, sizeof(ValueType) * matrixColCount);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not zero the Swap Vector x, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_b, b.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector b, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
	}

	// Preset the multiplyResult to zeros...
	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemset(device_multiplyResult, 0, sizeof(ValueType) * matrixRowCount);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not zero the multiply Result, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_nondeterministicChoiceIndices, nondeterministicChoiceIndices.data(), sizeof(IndexType) * (matrixColCount + 1), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector b, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
	}

#ifdef DEBUG
	std::cout << "(DLL) Finished copying data to GPU memory." << std::endl;
#endif

	// Data is on device, start Kernel
	while (!converged && iterationCount < maxIterationCount) { // In a sub-area since transfer of control via label evades initialization
		cusp::detail::device::storm_cuda_opt_spmv_csr_vector<ValueType>(matrixRowCount, matrixNnzCount, device_matrixRowIndices, device_matrixColIndicesAndValues, device_x, device_multiplyResult);
		CUDA_CHECK_ALL_ERRORS();

		thrust::device_ptr<ValueType> devicePtrThrust_b(device_b);
		thrust::device_ptr<ValueType> devicePtrThrust_multiplyResult(device_multiplyResult);

		// Transform: Add multiplyResult + b inplace to multiplyResult
		thrust::transform(devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + matrixRowCount, devicePtrThrust_b, devicePtrThrust_multiplyResult, thrust::plus<ValueType>());
		CUDA_CHECK_ALL_ERRORS();

		// Reduce: Reduce multiplyResult to a new x vector
		cusp::detail::device::storm_cuda_opt_vector_reduce<Minimize, ValueType>(matrixColCount, matrixRowCount, device_nondeterministicChoiceIndices, device_xSwap, device_multiplyResult);
		CUDA_CHECK_ALL_ERRORS();

		// Check for convergence
		// Transform: x = abs(x - xSwap)/ xSwap
		thrust::device_ptr<ValueType> devicePtrThrust_x(device_x);
		thrust::device_ptr<ValueType> devicePtrThrust_x_end(device_x + matrixColCount);
		thrust::device_ptr<ValueType> devicePtrThrust_xSwap(device_xSwap);
		thrust::transform(devicePtrThrust_x, devicePtrThrust_x_end, devicePtrThrust_xSwap, devicePtrThrust_x, equalModuloPrecision<ValueType, Relative>());
		CUDA_CHECK_ALL_ERRORS();

		// Reduce: get Max over x and check for res < Precision
		ValueType maxX = thrust::reduce(devicePtrThrust_x, devicePtrThrust_x_end, -std::numeric_limits<ValueType>::max(), thrust::maximum<ValueType>());
		CUDA_CHECK_ALL_ERRORS();
		converged = (maxX < precision);
		++iterationCount;

		// Swap pointers, device_x always contains the most current result
		std::swap(device_x, device_xSwap);
	}

	if (!converged && (iterationCount == maxIterationCount)) {
		iterationCount = 0;
		errorOccured = true;
	}

#ifdef DEBUG
	std::cout << "(DLL) Finished kernel execution." << std::endl;
	std::cout << "(DLL) Executed " << iterationCount << " of max. " << maxIterationCount << " Iterations." << std::endl;
#endif

	// Get x back from the device
	cudaCopyResult = hipMemcpy(x.data(), device_x, sizeof(ValueType) * matrixColCount, hipMemcpyDeviceToHost);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy back data for result vector x, Error Code " << cudaCopyResult << std::endl;
		errorOccured = true;
		goto cleanup;
	}

#ifdef DEBUG
	std::cout << "(DLL) Finished copying result data." << std::endl;
#endif

	// All code related to freeing memory and clearing up the device
cleanup:
	if (device_matrixRowIndices != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_matrixRowIndices);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Matrix Row Indices, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
		}
		device_matrixRowIndices = nullptr;
	}
	if (device_matrixColIndicesAndValues != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_matrixColIndicesAndValues);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Matrix Column Indices and Values, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
		}
		device_matrixColIndicesAndValues = nullptr;
	}
	if (device_x != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_x);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector x, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
		}
		device_x = nullptr;
	}
	if (device_xSwap != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_xSwap);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector x swap, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
		}
		device_xSwap = nullptr;
	}
	if (device_b != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_b);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector b, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
		}
		device_b = nullptr;
	}
	if (device_multiplyResult != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_multiplyResult);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector multiplyResult, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
		}
		device_multiplyResult = nullptr;
	}
	if (device_nondeterministicChoiceIndices != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_nondeterministicChoiceIndices);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Nondeterministic Choice Indices, Error Code " << cudaFreeResult << "." << std::endl;
			errorOccured = true;
		}
		device_nondeterministicChoiceIndices = nullptr;
	}
#ifdef DEBUG
	std::cout << "(DLL) Finished cleanup." << std::endl;
#endif

	return !errorOccured;
}

template <typename IndexType, typename ValueType>
void basicValueIteration_spmv(uint_fast64_t const matrixColCount, std::vector<IndexType> const& matrixRowIndices, std::vector<storm::storage::MatrixEntry<uint_fast64_t, ValueType>> const& columnIndicesAndValues, std::vector<ValueType> const& x, std::vector<ValueType>& b) {
	IndexType* device_matrixRowIndices = nullptr;
	ValueType* device_matrixColIndicesAndValues = nullptr;
	ValueType* device_x = nullptr;
	ValueType* device_multiplyResult = nullptr;

#ifdef DEBUG
	std::cout.sync_with_stdio(true);
	std::cout << "(DLL) Entering CUDA Function: basicValueIteration_spmv" << std::endl;
	std::cout << "(DLL) Device has " << getTotalCudaMemory() << " Bytes of Memory with " << getFreeCudaMemory() << "Bytes free (" << (static_cast<double>(getFreeCudaMemory()) / static_cast<double>(getTotalCudaMemory()))*100 << "%)." << std::endl; 
	size_t memSize = sizeof(IndexType) * matrixRowIndices.size() + sizeof(IndexType) * columnIndicesAndValues.size() * 2 + sizeof(ValueType) * x.size() + sizeof(ValueType) * b.size();
	std::cout << "(DLL) We will allocate " << memSize << " Bytes." << std::endl;
#endif

	const IndexType matrixRowCount = matrixRowIndices.size() - 1;
	const IndexType matrixNnzCount = columnIndicesAndValues.size();

	hipError_t cudaMallocResult;

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_matrixRowIndices), sizeof(IndexType) * (matrixRowCount + 1));
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Matrix Row Indices, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

#ifdef STORM_CUDAPLUGIN_HAVE_64BIT_FLOAT_ALIGNMENT
	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_matrixColIndicesAndValues), sizeof(IndexType) * matrixNnzCount + sizeof(IndexType) * matrixNnzCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Matrix Column Indices And Values, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}
#else
	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_matrixColIndicesAndValues), sizeof(IndexType) * matrixNnzCount + sizeof(ValueType) * matrixNnzCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Matrix Column Indices And Values, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}
#endif

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_x), sizeof(ValueType) * matrixColCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector x, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_multiplyResult), sizeof(ValueType) * matrixRowCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector multiplyResult, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

#ifdef DEBUG
	std::cout << "(DLL) Finished allocating memory." << std::endl;
#endif

	// Memory allocated, copy data to device
	hipError_t cudaCopyResult;

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_matrixRowIndices, matrixRowIndices.data(), sizeof(IndexType) * (matrixRowCount + 1), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Matrix Row Indices, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

#ifdef STORM_CUDAPLUGIN_HAVE_64BIT_FLOAT_ALIGNMENT
	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_matrixColIndicesAndValues, columnIndicesAndValues.data(), (sizeof(IndexType) * matrixNnzCount) + (sizeof(IndexType) * matrixNnzCount), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Matrix Column Indices and Values, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}
#else
	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_matrixColIndicesAndValues, columnIndicesAndValues.data(), (sizeof(IndexType) * matrixNnzCount) + (sizeof(ValueType) * matrixNnzCount), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Matrix Column Indices and Values, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}
#endif

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_x, x.data(), sizeof(ValueType) * matrixColCount, hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector x, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	// Preset the multiplyResult to zeros...
	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemset(device_multiplyResult, 0, sizeof(ValueType) * matrixRowCount);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not zero the multiply Result, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

#ifdef DEBUG
	std::cout << "(DLL) Finished copying data to GPU memory." << std::endl;
#endif

	cusp::detail::device::storm_cuda_opt_spmv_csr_vector<ValueType>(matrixRowCount, matrixNnzCount, device_matrixRowIndices, device_matrixColIndicesAndValues, device_x, device_multiplyResult);
	CUDA_CHECK_ALL_ERRORS();

#ifdef DEBUG
	std::cout << "(DLL) Finished kernel execution." << std::endl;
#endif

	// Get result back from the device
	cudaCopyResult = hipMemcpy(b.data(), device_multiplyResult, sizeof(ValueType) * matrixRowCount, hipMemcpyDeviceToHost);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy back data for result vector, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

#ifdef DEBUG
	std::cout << "(DLL) Finished copying result data." << std::endl;
#endif

	// All code related to freeing memory and clearing up the device
cleanup:
	if (device_matrixRowIndices != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_matrixRowIndices);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Matrix Row Indices, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_matrixRowIndices = nullptr;
	}
	if (device_matrixColIndicesAndValues != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_matrixColIndicesAndValues);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Matrix Column Indices and Values, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_matrixColIndicesAndValues = nullptr;
	}
	if (device_x != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_x);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector x, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_x = nullptr;
	}
	if (device_multiplyResult != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_multiplyResult);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector multiplyResult, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_multiplyResult = nullptr;
	}
#ifdef DEBUG
	std::cout << "(DLL) Finished cleanup." << std::endl;
#endif
}

template <typename ValueType>
void basicValueIteration_addVectorsInplace(std::vector<ValueType>& a, std::vector<ValueType> const& b) {
	ValueType* device_a = nullptr;
	ValueType* device_b = nullptr;

	const size_t vectorSize = std::max(a.size(), b.size());

	hipError_t cudaMallocResult;

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_a), sizeof(ValueType) * vectorSize);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector a, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_b), sizeof(ValueType) * vectorSize);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector b, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	// Memory allocated, copy data to device
	hipError_t cudaCopyResult;

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_a, a.data(), sizeof(ValueType) * vectorSize, hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector a, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_b, b.data(), sizeof(ValueType) * vectorSize, hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector b, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}
	
	do {
		// Transform: Add multiplyResult + b inplace to multiplyResult
		thrust::device_ptr<ValueType> devicePtrThrust_a(device_a);
		thrust::device_ptr<ValueType> devicePtrThrust_b(device_b);
		thrust::transform(devicePtrThrust_a, devicePtrThrust_a + vectorSize, devicePtrThrust_b, devicePtrThrust_a, thrust::plus<ValueType>());
		CUDA_CHECK_ALL_ERRORS();
	} while (false);

	// Get result back from the device
	cudaCopyResult = hipMemcpy(a.data(), device_a, sizeof(ValueType) * vectorSize, hipMemcpyDeviceToHost);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy back data for result vector, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	// All code related to freeing memory and clearing up the device
cleanup:
	if (device_a != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_a);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector a, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_a = nullptr;
	}
	if (device_b != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_b);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector b, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_b = nullptr;
	}
}

template <typename IndexType, typename ValueType, bool Minimize>
void basicValueIteration_reduceGroupedVector(std::vector<ValueType> const& groupedVector, std::vector<IndexType> const& grouping, std::vector<ValueType>& targetVector) {
	ValueType* device_groupedVector = nullptr;
	IndexType* device_grouping = nullptr;
	ValueType* device_target = nullptr;

	const size_t groupedSize = groupedVector.size();
	const size_t groupingSize = grouping.size();
	const size_t targetSize = targetVector.size();

	hipError_t cudaMallocResult;

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_groupedVector), sizeof(ValueType) * groupedSize);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector groupedVector, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_grouping), sizeof(IndexType) * groupingSize);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector grouping, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_target), sizeof(ValueType) * targetSize);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector targetVector, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	// Memory allocated, copy data to device
	hipError_t cudaCopyResult;

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_groupedVector, groupedVector.data(), sizeof(ValueType) * groupedSize, hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector groupedVector, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_grouping, grouping.data(), sizeof(IndexType) * groupingSize, hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector grouping, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}
	
	do {
		// Reduce: Reduce multiplyResult to a new x vector
		cusp::detail::device::storm_cuda_opt_vector_reduce<Minimize, ValueType>(groupingSize - 1, groupedSize, device_grouping, device_target, device_groupedVector);
		CUDA_CHECK_ALL_ERRORS();
	} while (false);

	// Get result back from the device
	cudaCopyResult = hipMemcpy(targetVector.data(), device_target, sizeof(ValueType) * targetSize, hipMemcpyDeviceToHost);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy back data for result vector, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	// All code related to freeing memory and clearing up the device
cleanup:
	if (device_groupedVector != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_groupedVector);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector groupedVector, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_groupedVector = nullptr;
	}
	if (device_grouping != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_grouping);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector grouping, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_grouping = nullptr;
	}
	if (device_target != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_target);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector target, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_target = nullptr;
	}
}

template <typename ValueType, bool Relative>
void basicValueIteration_equalModuloPrecision(std::vector<ValueType> const& x, std::vector<ValueType> const& y, ValueType& maxElement) {
	ValueType* device_x = nullptr;
	ValueType* device_y = nullptr;

	const size_t vectorSize = x.size();

	hipError_t cudaMallocResult;

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_x), sizeof(ValueType) * vectorSize);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector x, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_y), sizeof(ValueType) * vectorSize);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector y, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	// Memory allocated, copy data to device
	hipError_t cudaCopyResult;

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_x, x.data(), sizeof(ValueType) * vectorSize, hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector x, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_y, y.data(), sizeof(ValueType) * vectorSize, hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector y, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}
	
	do {
		// Transform: x = abs(x - xSwap)/ xSwap
		thrust::device_ptr<ValueType> devicePtrThrust_x(device_x);
		thrust::device_ptr<ValueType> devicePtrThrust_y(device_y);
		thrust::transform(devicePtrThrust_x, devicePtrThrust_x + vectorSize, devicePtrThrust_y, devicePtrThrust_x, equalModuloPrecision<ValueType, Relative>());
		CUDA_CHECK_ALL_ERRORS();

		// Reduce: get Max over x and check for res < Precision
		maxElement = thrust::reduce(devicePtrThrust_x, devicePtrThrust_x + vectorSize, -std::numeric_limits<ValueType>::max(), thrust::maximum<ValueType>());
		CUDA_CHECK_ALL_ERRORS();
	} while (false);

	// All code related to freeing memory and clearing up the device
cleanup:
	if (device_x != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_x);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector x, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_x = nullptr;
	}
	if (device_y != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_y);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector y, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_y = nullptr;
	}
}

/*
 * Declare and implement all exported functions for these Kernels here
 *
 */

void basicValueIteration_spmv_uint64_double(uint_fast64_t const matrixColCount, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<storm::storage::MatrixEntry<uint_fast64_t, double>> const& columnIndicesAndValues, std::vector<double> const& x, std::vector<double>& b) {
	basicValueIteration_spmv<uint_fast64_t, double>(matrixColCount, matrixRowIndices, columnIndicesAndValues, x, b);
}

void basicValueIteration_addVectorsInplace_double(std::vector<double>& a, std::vector<double> const& b) {
	basicValueIteration_addVectorsInplace<double>(a, b);
}

void basicValueIteration_reduceGroupedVector_uint64_double_minimize(std::vector<double> const& groupedVector, std::vector<uint_fast64_t> const& grouping, std::vector<double>& targetVector) {
	basicValueIteration_reduceGroupedVector<uint_fast64_t, double, true>(groupedVector, grouping, targetVector);
}

void basicValueIteration_reduceGroupedVector_uint64_double_maximize(std::vector<double> const& groupedVector, std::vector<uint_fast64_t> const& grouping, std::vector<double>& targetVector) {
	basicValueIteration_reduceGroupedVector<uint_fast64_t, double, false>(groupedVector, grouping, targetVector);
}

void basicValueIteration_equalModuloPrecision_double_Relative(std::vector<double> const& x, std::vector<double> const& y, double& maxElement) {
	basicValueIteration_equalModuloPrecision<double, true>(x, y, maxElement);
}

void basicValueIteration_equalModuloPrecision_double_NonRelative(std::vector<double> const& x, std::vector<double> const& y, double& maxElement) {
	basicValueIteration_equalModuloPrecision<double, false>(x, y, maxElement);
}

// Float
void basicValueIteration_spmv_uint64_float(uint_fast64_t const matrixColCount, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<storm::storage::MatrixEntry<uint_fast64_t, float>> const& columnIndicesAndValues, std::vector<float> const& x, std::vector<float>& b) {
	basicValueIteration_spmv<uint_fast64_t, float>(matrixColCount, matrixRowIndices, columnIndicesAndValues, x, b);
}

void basicValueIteration_addVectorsInplace_float(std::vector<float>& a, std::vector<float> const& b) {
	basicValueIteration_addVectorsInplace<float>(a, b);
}

void basicValueIteration_reduceGroupedVector_uint64_float_minimize(std::vector<float> const& groupedVector, std::vector<uint_fast64_t> const& grouping, std::vector<float>& targetVector) {
	basicValueIteration_reduceGroupedVector<uint_fast64_t, float, true>(groupedVector, grouping, targetVector);
}

void basicValueIteration_reduceGroupedVector_uint64_float_maximize(std::vector<float> const& groupedVector, std::vector<uint_fast64_t> const& grouping, std::vector<float>& targetVector) {
	basicValueIteration_reduceGroupedVector<uint_fast64_t, float, false>(groupedVector, grouping, targetVector);
}

void basicValueIteration_equalModuloPrecision_float_Relative(std::vector<float> const& x, std::vector<float> const& y, float& maxElement) {
	basicValueIteration_equalModuloPrecision<float, true>(x, y, maxElement);
}

void basicValueIteration_equalModuloPrecision_float_NonRelative(std::vector<float> const& x, std::vector<float> const& y, float& maxElement) {
	basicValueIteration_equalModuloPrecision<float, false>(x, y, maxElement);
}

bool basicValueIteration_mvReduce_uint64_double_minimize(uint_fast64_t const maxIterationCount, double const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<storm::storage::MatrixEntry<uint_fast64_t, double>> const& columnIndicesAndValues, std::vector<double>& x, std::vector<double> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, size_t& iterationCount) {
	if (relativePrecisionCheck) {
		return basicValueIteration_mvReduce<true, true, uint_fast64_t, double>(maxIterationCount, precision, matrixRowIndices, columnIndicesAndValues, x, b, nondeterministicChoiceIndices, iterationCount);
	} else {
		return basicValueIteration_mvReduce<true, false, uint_fast64_t, double>(maxIterationCount, precision, matrixRowIndices, columnIndicesAndValues, x, b, nondeterministicChoiceIndices, iterationCount);
	}
}

bool basicValueIteration_mvReduce_uint64_double_maximize(uint_fast64_t const maxIterationCount, double const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<storm::storage::MatrixEntry<uint_fast64_t, double>> const& columnIndicesAndValues, std::vector<double>& x, std::vector<double> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, size_t& iterationCount) {
	if (relativePrecisionCheck) {
		return basicValueIteration_mvReduce<false, true, uint_fast64_t, double>(maxIterationCount, precision, matrixRowIndices, columnIndicesAndValues, x, b, nondeterministicChoiceIndices, iterationCount);
	} else {
		return basicValueIteration_mvReduce<false, false, uint_fast64_t, double>(maxIterationCount, precision, matrixRowIndices, columnIndicesAndValues, x, b, nondeterministicChoiceIndices, iterationCount);
	}
}

bool basicValueIteration_mvReduce_uint64_float_minimize(uint_fast64_t const maxIterationCount, double const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<storm::storage::MatrixEntry<uint_fast64_t, float>> const& columnIndicesAndValues, std::vector<float>& x, std::vector<float> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, size_t& iterationCount) {
	if (relativePrecisionCheck) {
		return basicValueIteration_mvReduce<true, true, uint_fast64_t, float>(maxIterationCount, precision, matrixRowIndices, columnIndicesAndValues, x, b, nondeterministicChoiceIndices, iterationCount);
	} else {
		return basicValueIteration_mvReduce<true, false, uint_fast64_t, float>(maxIterationCount, precision, matrixRowIndices, columnIndicesAndValues, x, b, nondeterministicChoiceIndices, iterationCount);
	}
}

bool basicValueIteration_mvReduce_uint64_float_maximize(uint_fast64_t const maxIterationCount, double const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<storm::storage::MatrixEntry<uint_fast64_t, float>> const& columnIndicesAndValues, std::vector<float>& x, std::vector<float> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices, size_t& iterationCount) {
	if (relativePrecisionCheck) {
		return basicValueIteration_mvReduce<false, true, uint_fast64_t, float>(maxIterationCount, precision, matrixRowIndices, columnIndicesAndValues, x, b, nondeterministicChoiceIndices, iterationCount);
	} else {
		return basicValueIteration_mvReduce<false, false, uint_fast64_t, float>(maxIterationCount, precision, matrixRowIndices, columnIndicesAndValues, x, b, nondeterministicChoiceIndices, iterationCount);
	}
}

size_t basicValueIteration_mvReduce_uint64_double_calculateMemorySize(size_t const rowCount, size_t const rowGroupCount, size_t const nnzCount) {
	size_t const valueTypeSize = sizeof(double);
	size_t const indexTypeSize = sizeof(uint_fast64_t);

	/*
	IndexType* device_matrixRowIndices = nullptr;
	IndexType* device_matrixColIndices = nullptr;
	ValueType* device_matrixValues = nullptr;
	ValueType* device_x = nullptr;
	ValueType* device_xSwap = nullptr;
	ValueType* device_b = nullptr;
	ValueType* device_multiplyResult = nullptr;
	IndexType* device_nondeterministicChoiceIndices = nullptr;
	*/

	// Row Indices, Column Indices, Values, Choice Indices
	size_t const matrixDataSize = ((rowCount + 1) * indexTypeSize) + (nnzCount * indexTypeSize) + (nnzCount * valueTypeSize) + ((rowGroupCount + 1) * indexTypeSize);
	// Vectors x, xSwap, b, multiplyResult
	size_t const vectorSizes = (rowGroupCount * valueTypeSize) + (rowGroupCount * valueTypeSize) + (rowCount * valueTypeSize) + (rowCount * valueTypeSize);

	return (matrixDataSize + vectorSizes);
}

size_t basicValueIteration_mvReduce_uint64_float_calculateMemorySize(size_t const rowCount, size_t const rowGroupCount, size_t const nnzCount) {
	size_t const valueTypeSize = sizeof(float);
	size_t const indexTypeSize = sizeof(uint_fast64_t);

	/*
	IndexType* device_matrixRowIndices = nullptr;
	IndexType* device_matrixColIndices = nullptr;
	ValueType* device_matrixValues = nullptr;
	ValueType* device_x = nullptr;
	ValueType* device_xSwap = nullptr;
	ValueType* device_b = nullptr;
	ValueType* device_multiplyResult = nullptr;
	IndexType* device_nondeterministicChoiceIndices = nullptr;
	*/

	// Row Indices, Column Indices, Values, Choice Indices
	size_t const matrixDataSize = ((rowCount + 1) * indexTypeSize) + (nnzCount * indexTypeSize) + (nnzCount * valueTypeSize) + ((rowGroupCount + 1) * indexTypeSize);
	// Vectors x, xSwap, b, multiplyResult
	size_t const vectorSizes = (rowGroupCount * valueTypeSize) + (rowGroupCount * valueTypeSize) + (rowCount * valueTypeSize) + (rowCount * valueTypeSize);

	return (matrixDataSize + vectorSizes);
}